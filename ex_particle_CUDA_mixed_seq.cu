#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <limits.h>
#include <math.h>
#include <unistd.h>
#include <fcntl.h>
#include <float.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>

#define BLOCK_X 16
#define BLOCK_Y 16
#define PI 3.1415926535897932

/**
@var M value for Linear Congruential Generator (LCG); use GCC's value
 */
long M = INT_MAX;
/**
@var A value for LCG
 */
int A = 1103515245;
/**
@var C value for LCG
 */
int C = 12345;

/*****************************
 *GET_TIME
 *returns a long int representing the time
 *****************************/
long long get_time()
{
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return (tv.tv_sec * 1000000) + tv.tv_usec;
}

double elapsed_time(long long start_time, long long end_time)
{
    return (double)(end_time - start_time) / (1000 * 1000);
}

/*****************************
 * checkError
 * Checks for CUDA errors and prints them to the screen to help with
 * debugging of CUDA related programming
 *****************************/
void cudaCheck(hipError_t e)
{
    if (e != hipSuccess)
    {
        printf("\nCUDA error: %s\n", hipGetErrorString(e));
        exit(1);
    }
}

/****************************
CDF CALCULATE
CALCULATES CDF
param1 CDF
param2 weights
param3 nParticles
 *****************************/
__device__ void cdfCalc(double *CDF, double *weights, int nParticles)
{
    int x;
    CDF[0] = weights[0];
    for (x = 1; x < nParticles; x++)
    {
        CDF[x] = weights[x] + CDF[x - 1];
    }
}

/**
 * Generates a uniformly distributed random number using the provided seed and GCC's settings for the Linear Congruential Generator (LCG)
 * @see http://en.wikipedia.org/wiki/Linear_congruential_generator
 * @note This function is thread-safe
 * @param seed The seed array
 * @param index The specific index of the seed to be advanced
 * @return a uniformly distributed number [0, 1)
 */
double randu(int *seed, int index)
{
    int num = A * seed[index] + C;
    seed[index] = num % M;
    return fabs(seed[index] / ((double)M));
}

/**
 * Generates a normally distributed random number using the Box-Muller transformation
 * @note This function is thread-safe
 * @param seed The seed array
 * @param index The specific index of the seed to be advanced
 * @return a double representing random number generated using the Box-Muller algorithm
 * @see http://en.wikipedia.org/wiki/Normal_distribution, section computing value for normal random distribution
 */
double randn(int *seed, int index)
{
    /*Box-Muller algorithm*/
    double u = randu(seed, index);
    double v = randu(seed, index);
    double cosine = cos(2 * PI * v);
    double rt = -2 * log(u);
    return sqrt(rt) * cosine;
}

template <typename T>
__device__ T deviceRound(T value)
{
    int newValue = (int)value;
    return value - (T)newValue < ((T)0.5)
               ? newValue
               : newValue + 1;
}

/**
 * Takes in a double and returns an integer that approximates to that double
 * @return if the mantissa < .5 => return value < input value; else return value > input value
 */
double roundDouble(double value)
{
    int newValue = (int)(value);
    if (value - newValue < .5)
        return newValue;
    else
        return newValue++;
}

/**
 * Set values of the 3D array to a newValue if that value is equal to the testValue
 * @param testValue The value to be replaced
 * @param newValue The value to replace testValue with
 * @param array3D The image vector
 * @param dimX The x dimension of the frame
 * @param dimY The y dimension of the frame
 * @param dimZ The number of frames
 */
void setIf(int testValue, int newValue, unsigned char *array3D, int *dimX, int *dimY, int *dimZ)
{
    int x, y, z;
    for (x = 0; x < *dimX; x++)
    {
        for (y = 0; y < *dimY; y++)
        {
            for (z = 0; z < *dimZ; z++)
            {
                if (array3D[x * *dimY * *dimZ + y * *dimZ + z] == testValue)
                    array3D[x * *dimY * *dimZ + y * *dimZ + z] = newValue;
            }
        }
    }
}

/**
 * Sets values of 3D matrix using randomly generated numbers from a normal distribution
 * @param array3D The video to be modified
 * @param dimX The x dimension of the frame
 * @param dimY The y dimension of the frame
 * @param dimZ The number of frames
 * @param seed The seed array
 */
void addNoise(unsigned char *array3D, int *dimX, int *dimY, int *dimZ, int *seed)
{
    int x, y, z;
    for (x = 0; x < *dimX; x++)
    {
        for (y = 0; y < *dimY; y++)
        {
            for (z = 0; z < *dimZ; z++)
            {
                array3D[x * *dimY * *dimZ + y * *dimZ + z] = array3D[x * *dimY * *dimZ + y * *dimZ + z] + (unsigned char)(5 * randn(seed, 0));
            }
        }
    }
}

/**
 * Fills a radius x radius matrix representing the disk
 * @param disk The pointer to the disk to be made
 * @param radius  The radius of the disk to be made
 */
void strelDisk(int *disk, int radius)
{
    int diameter = radius * 2 - 1;
    int x, y;
    for (x = 0; x < diameter; x++)
    {
        for (y = 0; y < diameter; y++)
        {
            double distance = sqrt(pow((double)(x - radius + 1), 2) + pow((double)(y - radius + 1), 2));
            if (distance < radius)
                disk[x * diameter + y] = 1;
        }
    }
}

/**
 * Dilates the provided video
 * @param matrix The video to be dilated
 * @param posX The x location of the pixel to be dilated
 * @param posY The y location of the pixel to be dilated
 * @param poxZ The z location of the pixel to be dilated
 * @param dimX The x dimension of the frame
 * @param dimY The y dimension of the frame
 * @param dimZ The number of frames
 * @param error The error radius
 */
void dilate_matrix(unsigned char *matrix, int posX, int posY, int posZ, int dimX, int dimY, int dimZ, int error)
{
    int startX = posX - error;
    while (startX < 0)
        startX++;
    int startY = posY - error;
    while (startY < 0)
        startY++;
    int endX = posX + error;
    while (endX > dimX)
        endX--;
    int endY = posY + error;
    while (endY > dimY)
        endY--;
    int x, y;
    for (x = startX; x < endX; x++)
    {
        for (y = startY; y < endY; y++)
        {
            double distance = sqrt(pow((double)(x - posX), 2) + pow((double)(y - posY), 2));
            if (distance < error)
                matrix[x * dimY * dimZ + y * dimZ + posZ] = 1;
        }
    }
}

/**
 * Dilates the target matrix using the radius as a guide
 * @param matrix The reference matrix
 * @param dimX The x dimension of the video
 * @param dimY The y dimension of the video
 * @param dimZ The z dimension of the video
 * @param error The error radius to be dilated
 * @param newMatrix The target matrix
 */
void imdilate_disk(unsigned char *matrix, int dimX, int dimY, int dimZ, int error, unsigned char *newMatrix)
{
    int x, y, z;
    for (z = 0; z < dimZ; z++)
    {
        for (x = 0; x < dimX; x++)
        {
            for (y = 0; y < dimY; y++)
            {
                if (matrix[x * dimY * dimZ + y * dimZ + z] == 1)
                {
                    dilate_matrix(newMatrix, x, y, z, dimX, dimY, dimZ, error);
                }
            }
        }
    }
}

/**
 * Fills a 2D array describing the offsets of the disk object
 * @param se The disk object
 * @param numOnes The number of ones in the disk
 * @param neighbors The array that will contain the offsets
 * @param radius The radius used for dilation
 */
void getneighbors(int *se, int numOnes, int *neighbors, int radius)
{
    int x, y;
    int neighY = 0;
    int center = radius - 1;
    int diameter = radius * 2 - 1;
    for (x = 0; x < diameter; x++)
    {
        for (y = 0; y < diameter; y++)
        {
            if (se[x * diameter + y])
            {
                neighbors[neighY * 2] = (int)(y - center);
                neighbors[neighY * 2 + 1] = (int)(x - center);
                neighY++;
            }
        }
    }
}

/**
 * The synthetic video sequence we will work with here is composed of a
 * single moving object, circular in shape (fixed radius)
 * The motion here is a linear motion
 * the foreground intensity and the background intensity is known
 * the image is corrupted with zero mean Gaussian noise
 * @param I The video itself
 * @param IszX The x dimension of the video
 * @param IszY The y dimension of the video
 * @param Nfr The number of frames of the video
 * @param seed The seed array used for number generation
 */
void videoSequence(unsigned char *I, int IszX, int IszY, int Nfr, int *seed)
{
    int k;
    int max_size = IszX * IszY * Nfr;
    /*get object centers*/
    int x0 = (int)roundDouble(IszY / 2.0);
    int y0 = (int)roundDouble(IszX / 2.0);
    I[x0 * IszY * Nfr + y0 * Nfr + 0] = 1;

    FILE *fp = fopen("movement_true.txt", "w");
    fprintf(fp, "TRUE MOVEMENT: \n");

    /*move point*/
    int xk, yk, pos;
    for (k = 1; k < Nfr; k++)
    {
        xk = abs(x0 + (k - 1));
        yk = abs(y0 - 2 * (k - 1));
        fprintf(fp, ". <%d, %d>\n", xk, yk);
        pos = yk * IszY * Nfr + xk * Nfr + k;
        if (pos >= max_size)
            pos = 0;
        I[pos] = 1;
    }

    fprintf(fp, "end\n");

    /*dilate matrix*/
    unsigned char *newMatrix = (unsigned char *)malloc(sizeof(unsigned char) * IszX * IszY * Nfr);
    imdilate_disk(I, IszX, IszY, Nfr, 5, newMatrix);
    int x, y;
    for (x = 0; x < IszX; x++)
    {
        for (y = 0; y < IszY; y++)
        {
            for (k = 0; k < Nfr; k++)
            {
                I[x * IszY * Nfr + y * Nfr + k] = newMatrix[x * IszY * Nfr + y * Nfr + k];
            }
        }
    }
    free(newMatrix);

    /*define background, add noise*/
    setIf(0, 100, I, &IszX, &IszY, &Nfr);
    setIf(1, 228, I, &IszX, &IszY, &Nfr);
    /*add noise*/
    addNoise(I, &IszX, &IszY, &Nfr, seed);
}

/**
 * A set of inline overload forwarders.
 * Gsqrt short for generic square root.
 */
__device__ inline nv_half Gsqrt(nv_half x) { return hsqrt(x); }
__device__ inline float Gsqrt(float x) { return sqrtf(x); }
__device__ inline double Gsqrt(double x) { return sqrt(x); }

/**
 * A set of inline overload forwarders.
 * Gexp short for generic square root.
 */
__device__ inline nv_half Gexp(nv_half x) { return hexp(x); }
__device__ inline float Gexp(float x) { return exp(x); }
__device__ inline double Gexp(double x) { return exp(x); }

__device__ inline void h2print(half2 x) { printf("(%10.4f, %10.4f)\n", __low2float(x), __high2float(x)); }

/**
 * NVIDIA shared memory doesn't support unspecialized
 * function template. Below are the work around. So that
 * we don't have unspecialized initialization.
 *
 */
template <typename T>
struct SharedMemory
{
    // Ensure that we won't compile any un-specialized types
    __device__ T *getPointer()
    {
        extern __device__ void error(void);
        error();
        return NULL;
    }
};

template <>
struct SharedMemory<half>
{
    __device__ half *getPointer()
    {
        extern __shared__ half s_half[];
        return s_half;
    }
};

template <>
struct SharedMemory<float>
{
    __device__ float *getPointer()
    {
        extern __shared__ float s_float[];
        return s_float;
    }
};

template <>
struct SharedMemory<double>
{
    __device__ double *getPointer()
    {
        extern __shared__ double s_double[];
        return s_double;
    }
};

/**
 * Initialize cuRAND state for random number generation.
 * Every threads received the same seed, but different sequence.
 * @param states The hiprandState array for each threads.
 * @param nStates The number of states
 * @param seed The generator seed
 */
__global__ void curandSetupKernel(hiprandState *states, int nStates, int seed)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < nStates)
        hiprand_init(seed, idx, 0, &states[idx]);
}

/**
 * Propagate all particles to the next steps.
 * Sampling from true distribution and all model
 * parameters are assumed to be known.
 * @param states the cuRAND state arrays for each threads
 * @param X the X coordinates of current particles
 * @param Y the Y coordinates of current particles
 * @param Ax the X coordinates of the selected ancestors
 * @param Ay the Y coordinates of the selected ancestors
 */
template <typename T>
__global__ void propagationKernel(hiprandState *states, T *X, T *Y, T *Ax, T *Ay, int nParticles)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    hiprandState localState = states[idx];

    T sigmaX = 5.0;
    T sigmaY = 2.0;
    T muX = 1.0;
    T muY = 2.0;

    if (idx < nParticles)
    {
        X[idx] = Ax[idx] + muX + sigmaX * (T)hiprand_normal(&localState);
        Y[idx] = Ay[idx] - muY + sigmaY * (T)hiprand_normal(&localState);
    }

    states[idx] = localState;
}

/**
 * Evaluate the likelihood of each particles
 * giving the observation at frame T.
 * @param X
 * @param Y
 * @param nParticles
 * @param objXy
 * @param countOnes
 * @param likelihood
 * @param I
 * @param IszY
 * @param Nfr
 * @param maxSize
 */
template <typename T>
__global__ void likelihoodKernel(T *X, T *Y, int nParticles, int *objXy, int countOnes, T *likelihood, unsigned char *I, int IszY, int Nfr, int maxSize, int frIdx)
{
    int i, indX, indY;

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int particleIdx = idx / countOnes;
    int pixelIdx = idx % countOnes;

    T fg, bg, local;

    if (pixelIdx == 0)
    {
        likelihood[particleIdx] = 0;
    }

    __syncthreads();

    if (particleIdx < nParticles)
    {
        indX = (int)deviceRound(X[particleIdx]) + objXy[pixelIdx * 2];
        indY = (int)deviceRound(Y[particleIdx]) + objXy[pixelIdx * 2 + 1];
        i = abs(indY * IszY * Nfr + indX * Nfr + frIdx);
        i = i < maxSize ? i : 0;
        bg = (T)(I[i] - 100) / Gsqrt((T)countOnes * (T)50.0);
        fg = (T)(I[i] - 228) / Gsqrt((T)countOnes * (T)50.0);
        local = bg * bg - fg * fg;

        atomicAdd(&likelihood[particleIdx], local);
    }
}

template <>
__global__ void likelihoodKernel(half *X, half *Y, int nParticles, int *objXy, int countOnes, half *likelihood, unsigned char *I, int IszY, int Nfr, int maxSize, int frIdx)
{
    int indX1, indY1, indX2, indY2, ind1, ind2;

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int particleIdx2 = idx / countOnes;
    int pixelIdx = idx % countOnes;
    int nParticles2 = nParticles / 2;

    half2 bg, fg, indX, indY;
    half2 denom = h2rsqrt(__half2half2((half)(countOnes * 50)));

    half2 *X2 = (half2 *)X, *Y2 = (half2 *)Y;
    half2 *likelihood2 = (half2 *)likelihood;

    if (particleIdx2 < nParticles2 && pixelIdx == 0)
    {
        likelihood2[particleIdx2] = __half2half2(0.0);
    }

    __syncthreads();

    if (particleIdx2 < nParticles2)
    {
        indX = X2[particleIdx2];
        indY = Y2[particleIdx2];

        indX1 = __half2int_rn(__low2half(indX)) + objXy[pixelIdx * 2];
        indY1 = __half2int_rn(__low2half(indY)) + objXy[pixelIdx * 2 + 1];
        indX2 = __half2int_rn(__high2half(indX)) + objXy[pixelIdx * 2];
        indY2 = __half2int_rn(__high2half(indY)) + objXy[pixelIdx * 2 + 1];

        ind1 = abs(indY1 * IszY * Nfr + indX1 * Nfr + frIdx);
        ind2 = abs(indY2 * IszY * Nfr + indX2 * Nfr + frIdx);
        ind1 = ind1 < maxSize ? ind1 : 0;
        ind2 = ind2 < maxSize ? ind2 : 0;

        bg = __halves2half2(__int2half_rn(I[ind1] - 100), __int2half_rn(I[ind2] - 100)) * denom;
        fg = __halves2half2(__int2half_rn(I[ind1] - 228), __int2half_rn(I[ind2] - 228)) * denom;

        bg = __hmul2(bg, bg);
        fg = __hmul2(fg, fg);

        atomicAdd(&likelihood2[particleIdx2], __hsub2(bg, fg));
    }
}

/**
 * Find the maximum value in the array.
 * Required for log-sum-exp tricks.
 * @param array
 * @param result
 * @param n
 */
template <typename T>
__global__ void findMaxKernel(T *array, T *result, int n)
{
    int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int localIdx = threadIdx.x;

    extern __shared__ double fmbuffer[];

    if (globalIdx < n)
    {
        fmbuffer[localIdx] = (double)array[globalIdx];
    }
    else
    {
        fmbuffer[localIdx] = -100;
    }

    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s /= 2)
    {
        if (localIdx < s)
        {
            double a = fmbuffer[localIdx];
            double b = fmbuffer[localIdx + s];
            fmbuffer[localIdx] = a > b ? a : b;
        }

        __syncthreads();
    }

    if (localIdx == 0)
    {
        result[blockIdx.x] = (T)fmbuffer[0];
    }

    __syncthreads();

    if (globalIdx == 0)
    {
        for (int i = 1; i < gridDim.x; i++)
            result[0] = result[0] > result[i] ? result[0] : result[i];
    }

    __syncthreads();
}

/**
 * Recalculate particle weights and normalize them.
 * This function operate on higher-precision type
 * to preserve numerical stability.
 * @note This function requires double precision
 * @param likelihood The particles likelihood
 * @param weights The particles weights
 * @param sum The reference to sum on global mem
 * @param nParticles The number of particles
 */
template <typename T>
__global__ void weightingKernel(T *likelihood, T *weights, T *cdf, T *sum, int nParticles, T *maxLikelihood)
{
    int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int localIdx = threadIdx.x;
    T unnormalized;

    SharedMemory<T> sharedMem;
    T *buffer = sharedMem.getPointer();

    if (globalIdx < nParticles)
    {
        unnormalized = weights[globalIdx] * exp(likelihood[globalIdx] - maxLikelihood[0]);
        buffer[localIdx] = unnormalized;
    }
    else
    {
        buffer[localIdx] = 0.0;
    }

    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (threadIdx.x < s)
            buffer[localIdx] += buffer[localIdx + s];
        __syncthreads();
    }

    if (globalIdx == 0)
    {
        *sum = 0;
    }

    __syncthreads();

    if (localIdx == 0)
    {
        atomicAdd(sum, buffer[0]);
    }

    __syncthreads();

    weights[globalIdx] = unnormalized / (*sum);

    if (globalIdx == 0)
    {
        int x;
        cdf[0] = weights[0];
        for (x = 1; x < nParticles; x++)
        {
            cdf[x] = weights[x] + cdf[x - 1];
        }
    }

    __syncthreads();
}

template <>
__global__ void weightingKernel(half *likelihood, half *weights, half *cdf, half *sum, int nParticles, half *maxLikelihood)
{
    int nParticles2 = nParticles / 2;
    int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int localIdx = threadIdx.x;

    half2 unnormalized2;
    half2 *cdf2 = (half2 *)cdf;
    half2 *weights2 = (half2 *)weights;
    half2 *likelihood2 = (half2 *)likelihood;
    half2 maxLikelihood2 = __half2half2(maxLikelihood[0]);

    extern __shared__ half buffer[];
    half2 *buffer2 = (half2 *)buffer; 

    if (globalIdx < nParticles2)
    {
        unnormalized2 = weights2[globalIdx] * h2exp(__hsub2(likelihood2[globalIdx], maxLikelihood2));
        buffer2[localIdx] = unnormalized2;
    }
    else
    {
        buffer2[localIdx] = __half2half2(0.0);
    }

    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (threadIdx.x < s)
            buffer2[localIdx] += buffer2[localIdx + s];
        __syncthreads();
    }

    if (globalIdx == 0)
    {
        *sum = 0;
    }

    __syncthreads();

    if (localIdx == 0)
    {
        atomicAdd(sum, buffer[0] + buffer[1]);
    }

    __syncthreads();

    if (globalIdx < nParticles2) {
        weights2[globalIdx] = unnormalized2 / __half2half2(*sum);
    }

    if (globalIdx == 0)
    {
        half2 temp, prev = __half2half2(0.0); 
        for (int x = 0; x < nParticles2; x++)
        {
            temp = weights2[x]; 
            temp += __halves2half2(0.0, __low2half(temp)); 
            cdf2[x] = temp + prev; 
            prev += __half2half2(__high2half(temp)); 
        }
    }

    __syncthreads();
}

/**
 * Resamples particles from a set of selected
 * ancestors using a systematic resmapling scheme.
 * @param states cuRAND states for each threads
 * @param X The X coordinate of current particles
 * @param Y The Y coordinate of current particles
 * @param Ax The X coordinate of ancestors for next gen
 * @param Ax The Y coordinate of ancestors for next gen
 * @param cdf The CDF of normalized weights to resample from
 * @param u The uniform random variable needed for resampling
 * @param nParticles The number of particles in the simulation
 */
template <typename T>
__global__ void resamplingKernel(hiprandState *states, T *X, T *Y, T *Ax, T *Ay, T *weights, T *cdf, T *u, int nParticles)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    T multiplier = cdf[nParticles - 1];

    if (idx == 0)
    {
        hiprandState localState = states[idx];
        u[0] = (T)hiprand_uniform(&localState) * multiplier / (T)nParticles;
        states[idx] = localState;
    }

    __syncthreads();

    if (idx != 0 && idx < nParticles)
    {
        u[idx] = u[0] + (T)idx * multiplier / (T)nParticles;
    }

    if (idx < nParticles)
    {
        int ancestor = -1;
        int x;

        for (x = 0; x < nParticles; x++)
        {
            if (cdf[x] >= u[idx])
            {
                ancestor = x;
                break;
            }
        }

        if (ancestor == -1)
        {
            ancestor = nParticles - 1;
        }

        Ax[idx] = X[ancestor];
        Ay[idx] = Y[ancestor];
        weights[idx] = (T)1 / (T)nParticles;
    }
}

template <>
__global__ void resamplingKernel(hiprandState *states, half *X, half *Y, half *Ax, half *Ay, half *weights, half *cdf, half *u, int nParticles)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int nParticles2 = nParticles / 2; 

    half m = cdf[nParticles - 1];
    half n = __int2half_rn(nParticles); 
    half q = m / n; 

    half2 m2 = __half2half2(m);
    half2 n2 = __half2half2(n);
    half2 q2 = m2 / n2; 

    half2 *weights2 = (half2 *)weights; 
    half2 *u2 = (half2 *)u;

    if (idx == 0)
    {
        hiprandState localState = states[idx];
        half urv = __double2half(hiprand_uniform(&localState)) * q;
        u2[0] = __halves2half2(urv, urv + q); 
        states[idx] = localState;
    }

    __syncthreads();

    if (idx != 0 && idx < nParticles2)
    {
        half2 temp = __halves2half2(2 * idx, 2 * idx + 1); 
        u2[idx] = __low2half2(u2[0]) + temp * q2;
    }

    if (idx < nParticles2) {
        weights2[idx] = h2rcp(n2);
    }

    __syncthreads(); 


    if (idx < nParticles2)
    {
        half2 res, unmoved, urv2 = u2[idx];
        bool allmoved;  
        int ancestorA = -1;
        int ancestorB = -1; 
        int x;

        for (x = 0; x < nParticles; x++)
        {
            res = __hge2(__half2half2(cdf[x]), urv2); 
            unmoved = __heq2(__half2half2(-1), __floats2half2_rn(ancestorA, ancestorB)); 
            allmoved = __hbne2(__half2half2(-1), __floats2half2_rn(ancestorA, ancestorB)); 
            
            if (__low2half(unmoved) && __low2half(res)) {
                ancestorA = x; 
            }

            if (__high2half(unmoved) && __high2half(res)) {
                ancestorB = x; 
            }

            if (allmoved) {
                break; 
            }
        }

        if (ancestorA == -1)
            ancestorA = nParticles - 1;

        if (ancestorB == -1)
            ancestorB = nParticles - 1; 

        Ax[idx] = X[ancestorA];
        Ay[idx] = Y[ancestorA];
        Ax[idx + 1] = X[ancestorB];
        Ay[idx + 1] = Y[ancestorB];
    }
}

/**
 * Calculate the kernel launch config for LL Kernel
 * based on the data type. When using half-precision
 * the number of threads can be reduced by half.
 * @param tpb the number of threads per block
 */
template <typename T>
inline int calcNumBlocks(int n, int m, int tpb) { return (n * m - 1) / tpb + 1; }
template <>
inline int calcNumBlocks<half>(int n, int m, int tpb) { return (n * m / 2 - 1) / tpb + 1; }

template <typename T>
inline int calcSharedMem(int tpb) { return tpb * sizeof(T); }
template <>
inline int calcSharedMem<half>(int tpb) { return tpb * 2 * sizeof(half); }

/**
 * The implementation of the particle filter using OpenMP for many frames
 * @see http://openmp.org/wp/
 * @note This function is designed to work with a video of several frames. In addition, it references a provided MATLAB function which takes the video, the objxy matrix and the x and y arrays as arguments and returns the likelihoods
 * @param I The video to be run
 * @param IszX The x dimension of the video
 * @param IszY The y dimension of the video
 * @param Nfr The number of frames
 * @param seed The seed array used for random number generation
 * @param nParticles The number of particles to be used
 */
template <typename T>
void particleFilter(unsigned char *I, int IszX, int IszY, int Nfr, int seed, int nParticles)
{
    hiprandState *states;
    int threadsPerBlocks = 256;
    int numBlocks = (nParticles - 1) / threadsPerBlocks + 1;
    int totalThreads = numBlocks * threadsPerBlocks;

    /**
     * Initial setup for the particle filter.
     */
    int maxSize = IszX * IszY * Nfr;
    T xe = (T)roundDouble(IszY / 2.0);
    T ye = (T)roundDouble(IszX / 2.0);

    /**
     * Target object template needed for
     * likelihood calculation
     */
    int radius = 5;
    int diameter = radius * 2 - 1;
    int *disk = (int *)malloc(diameter * diameter * sizeof(int));
    strelDisk(disk, radius);

    /**
     * Object template preprocessing.
     * Counting entries with one.
     */
    int x, y;
    int countOnes = 0;
    for (x = 0; x < diameter; x++)
    {
        for (y = 0; y < diameter; y++)
        {
            if (disk[x * diameter + y] == 1)
                countOnes++;
        }
    }

    /**
     * Collecting indices around the center
     * of object to test against.
     */
    int *objxy = (int *)malloc(countOnes * 2 * sizeof(int));
    getneighbors(disk, countOnes, objxy, radius);

    /**
     * Variables allocations ranging from
     * ancestorsXY, currentXY, uniformRV,
     * weights, likelihood, indices along with
     * device copies.
     */
    T *Ax = (T *)malloc(nParticles * sizeof(T));
    T *Ay = (T *)malloc(nParticles * sizeof(T));
    T *X = (T *)malloc(nParticles * sizeof(T));
    T *Y = (T *)malloc(nParticles * sizeof(T));
    T *U = (T *)malloc(nParticles * sizeof(T));

    T *cdf = (T *)malloc(nParticles * sizeof(T));
    T *weights = (T *)malloc(nParticles * sizeof(T));
    T *likelihood = (T *)malloc(nParticles * sizeof(T));

    T *deviceAx, *deviceAy, *deviceX, *deviceY, *deviceU;
    T *deviceLikelihood, *deviceCdf, *deviceWeights;
    T *deviceMaxLikelihood;
    T *deviceSum;

    int *deviceObjXy;
    unsigned char *deviceI;

    // Uniformly initialize weights
    for (x = 0; x < nParticles; x++)
    {
        weights[x] = (T)(1 / (double)nParticles);
    }

    // All particles begins at the center
    for (x = 0; x < nParticles; x++)
    {
        Ax[x] = (T)xe;
        Ay[x] = (T)ye;
    }

    /**
     * Memory allocations for various
     * variables listed above.
     */
    cudaCheck(hipMalloc(&deviceAx, nParticles * sizeof(T)));
    cudaCheck(hipMalloc(&deviceAy, nParticles * sizeof(T)));
    cudaCheck(hipMalloc(&deviceX, nParticles * sizeof(T)));
    cudaCheck(hipMalloc(&deviceY, nParticles * sizeof(T)));
    cudaCheck(hipMalloc(&deviceU, nParticles * sizeof(T)));

    cudaCheck(hipMalloc(&deviceSum, sizeof(T)));
    cudaCheck(hipMalloc(&deviceMaxLikelihood, numBlocks * sizeof(T)));
    cudaCheck(hipMalloc(&deviceCdf, nParticles * sizeof(T)));
    cudaCheck(hipMalloc(&deviceWeights, nParticles * sizeof(T)));
    cudaCheck(hipMalloc(&deviceObjXy, 2 * countOnes * sizeof(int)));
    cudaCheck(hipMalloc(&deviceI, IszX * IszY * Nfr * sizeof(unsigned char)));
    cudaCheck(hipMalloc(&states, totalThreads * sizeof(hiprandState)));

    cudaCheck(hipMalloc(&deviceLikelihood, nParticles * sizeof(T)));
    cudaCheck(hipMemset(deviceLikelihood, 0, nParticles * sizeof(T)));

    /**
     * Memory copy from host to device
     * for various variables.
     */
    int r;
    long long sendStart = get_time();
    cudaCheck(hipMemcpy(deviceI, I, IszX * IszY * Nfr * sizeof(unsigned char), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(deviceObjXy, objxy, 2 * countOnes * sizeof(int), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(deviceWeights, weights, nParticles * sizeof(T), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(deviceAx, Ax, nParticles * sizeof(T), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(deviceAy, Ay, nParticles * sizeof(T), hipMemcpyHostToDevice));
    cudaCheck(hipDeviceSynchronize());
    long long sendEnd = get_time();

    curandSetupKernel<<<numBlocks, threadsPerBlocks>>>(states, totalThreads, seed);

#ifdef TRACE
    FILE *fp = fopen("mixed_precision_trace.txt", "w");
    FILE *pfp = fopen("movement_hat.txt", "w");
    fprintf(pfp, "PREDICTED MOVEMENT:\n");
#endif

    long long kernelStart = get_time();
    for (r = 0; r < Nfr; r++)
    {
        propagationKernel<<<numBlocks, threadsPerBlocks>>>(states,
                                                           deviceX, deviceY,
                                                           deviceAx, deviceAy,
                                                           nParticles);

        int likelihoodBlocks = calcNumBlocks<T>(nParticles, countOnes, threadsPerBlocks);
        likelihoodKernel<<<likelihoodBlocks, threadsPerBlocks>>>(deviceX, deviceY,
                                                                 nParticles, deviceObjXy, countOnes,
                                                                 deviceLikelihood, deviceI,
                                                                 IszY, Nfr, maxSize, r);

        findMaxKernel<<<numBlocks, threadsPerBlocks, threadsPerBlocks * sizeof(double)>>>(deviceLikelihood,
                                                                                          deviceMaxLikelihood, nParticles);

        int weightingBlocks = calcNumBlocks<T>(nParticles, 1, threadsPerBlocks);
        int weightingSharedMem = calcSharedMem<T>(threadsPerBlocks);
        weightingKernel<<<weightingBlocks, threadsPerBlocks, weightingSharedMem>>>(deviceLikelihood,
                                                                                   deviceWeights, deviceCdf,
                                                                                   deviceSum, nParticles,
                                                                                   deviceMaxLikelihood);

#ifdef TRACE
        cudaCheck(hipMemcpy(likelihood, deviceLikelihood, nParticles * sizeof(T), hipMemcpyDeviceToHost));
        cudaCheck(hipMemcpy(weights, deviceWeights, nParticles * sizeof(T), hipMemcpyDeviceToHost));
        cudaCheck(hipMemcpy(cdf, deviceCdf, nParticles * sizeof(T), hipMemcpyDeviceToHost));
        cudaCheck(hipMemcpy(Ax, deviceAx, nParticles * sizeof(T), hipMemcpyDeviceToHost));
        cudaCheck(hipMemcpy(Ay, deviceAy, nParticles * sizeof(T), hipMemcpyDeviceToHost));
        cudaCheck(hipMemcpy(X, deviceX, nParticles * sizeof(T), hipMemcpyDeviceToHost));
        cudaCheck(hipMemcpy(Y, deviceY, nParticles * sizeof(T), hipMemcpyDeviceToHost));
        cudaCheck(hipMemcpy(U, deviceU, nParticles * sizeof(T), hipMemcpyDeviceToHost));

        fprintf(fp, "\nFRAME %d\n", r);
        for (int k = 0; k < nParticles; k++)
        {
            fprintf(fp, "<%5.2f, %5.2f> || <%5.2f, %5.2f> ~ %10.4f ~ %10.4f ^+ %10.4f ~ %10.4f \n",
                    (double)Ax[k], (double)Ay[k], (double)X[k], (double)Y[k],
                    (double)likelihood[k], (double)weights[k],
                    (double)cdf[k], (double)U[k]);
        }

        double xHat = 0.0;
        double yHat = 0.0;

        for (int k = 0; k < nParticles; k++)
        {
            xHat += (double)weights[k] * (double)X[k];
            yHat += (double)weights[k] * (double)Y[k];
        }

        fprintf(fp, "END FRAME %d ~ Predicted Position = <%f,%f>\n",
                r, xHat, yHat);
        fprintf(pfp, ". <%.2f, %.2f>\n", xHat, yHat);
#endif

        int resamplingBlocks = calcNumBlocks<T>(nParticles, 1, threadsPerBlocks);
        resamplingKernel<<<resamplingBlocks, threadsPerBlocks>>>(states,
                                                                 deviceX, deviceY,
                                                                 deviceAx, deviceAy,
                                                                 deviceWeights, deviceCdf,
                                                                 deviceU, nParticles);
    }

#ifdef TRACE
    fprintf(pfp, "end");
    fclose(fp);
    fclose(pfp);
#endif

    cudaCheck(hipDeviceSynchronize());

    long long kernelEnd = get_time();
    cudaCheck(hipMemcpy(likelihood, deviceLikelihood, nParticles * sizeof(T), hipMemcpyDeviceToHost));
    cudaCheck(hipMemcpy(weights, deviceWeights, nParticles * sizeof(T), hipMemcpyDeviceToHost));
    cudaCheck(hipMemcpy(cdf, deviceCdf, nParticles * sizeof(T), hipMemcpyDeviceToHost));
    cudaCheck(hipMemcpy(X, deviceX, nParticles * sizeof(T), hipMemcpyDeviceToHost));
    cudaCheck(hipMemcpy(Y, deviceY, nParticles * sizeof(T), hipMemcpyDeviceToHost));
    cudaCheck(hipDeviceSynchronize());
    long long freeTime = get_time();

    /**
     * Memory allocations for various
     * variables listed above.
     */
    cudaCheck(hipFree(deviceAx));
    cudaCheck(hipFree(deviceAy));
    cudaCheck(hipFree(deviceX));
    cudaCheck(hipFree(deviceY));
    cudaCheck(hipFree(deviceU));

    cudaCheck(hipFree(deviceCdf));
    cudaCheck(hipFree(deviceWeights));
    cudaCheck(hipFree(deviceObjXy));
    cudaCheck(hipFree(deviceSum));
    cudaCheck(hipFree(deviceI));
    cudaCheck(hipFree(deviceLikelihood));
    cudaCheck(hipFree(deviceMaxLikelihood));
    cudaCheck(hipFree(states));

    free(disk);
    free(objxy);
    free(X);
    free(Y);
    free(Ax);
    free(Ay);
    free(U);
    free(cdf);
    free(weights);
    free(likelihood);

    cudaCheck(hipDeviceSynchronize());
}

int main(int argc, char *argv[])
{

    const char *usage = "double.out -x <dimX> -y <dimY> -z <Nfr> -np <nParticles> -pcs <double|float|half> -seed <seed>";
    // check number of arguments
    if (argc != 13)
    {
        printf("%s\n", usage);
        return 0;
    }
    // check args deliminators
    if (strcmp(argv[1], "-x") || strcmp(argv[3], "-y") || strcmp(argv[5], "-z") || strcmp(argv[7], "-np") || strcmp(argv[9], "-pcs") || strcmp(argv[11], "-seed"))
    {
        printf("%s\n", usage);
        return 0;
    }

    int IszX, IszY, Nfr, nParticles, pcs, deviceSeed;

    // converting a string to a integer
    if (sscanf(argv[2], "%d", &IszX) == EOF)
    {
        printf("ERROR: dimX input is incorrect");
        return 0;
    }

    if (IszX <= 0)
    {
        printf("dimX must be > 0\n");
        return 0;
    }

    // converting a string to a integer
    if (sscanf(argv[4], "%d", &IszY) == EOF)
    {
        printf("ERROR: dimY input is incorrect");
        return 0;
    }

    if (IszY <= 0)
    {
        printf("dimY must be > 0\n");
        return 0;
    }

    // converting a string to a integer
    if (sscanf(argv[6], "%d", &Nfr) == EOF)
    {
        printf("ERROR: Number of frames input is incorrect");
        return 0;
    }

    if (Nfr <= 0)
    {
        printf("number of frames must be > 0\n");
        return 0;
    }

    // converting a string to a integer
    if (sscanf(argv[8], "%d", &nParticles) == EOF)
    {
        printf("ERROR: Number of particles input is incorrect");
        return 0;
    }

    if (nParticles <= 0)
    {
        printf("Number of particles must be > 0\n");
        return 0;
    }

    if (sscanf(argv[12], "%d", &deviceSeed) == EOF)
    {
        printf("ERROR: Seed is invalid");
        return 0;
    }

    if (!strcmp(argv[10], "half"))
    {
        pcs = 0;
    }
    else if (!strcmp(argv[10], "float"))
    {
        pcs = 1;
    }
    else if (!strcmp(argv[10], "double"))
    {
        pcs = 2;
    }
    else
    {
        printf("%s\n", usage);
        return 0;
    }

    // establish seed
    int *seed = (int *)malloc(sizeof(int) * nParticles);
    int i;
    for (i = 0; i < nParticles; i++)
        seed[i] = time(0) * i;
    // malloc matrix
    unsigned char *I = (unsigned char *)malloc(sizeof(unsigned char) * IszX * IszY * Nfr);

    long long start = get_time();
    videoSequence(I, IszX, IszY, Nfr, seed);
    long long endVideoSequence = get_time();
    printf("VIDEO SEQUENCE TOOK %f\n", elapsed_time(start, endVideoSequence));

    switch (pcs)
    {
    case 0:
        particleFilter<half>(I, IszX, IszY, Nfr, deviceSeed, nParticles);
        break;
    case 1:
        particleFilter<float>(I, IszX, IszY, Nfr, deviceSeed, nParticles);
        break;
    case 2:
        particleFilter<double>(I, IszX, IszY, Nfr, deviceSeed, nParticles);
        break;
    }

    long long endParticleFilter = get_time();
    printf("PARTICLE FILTER TOOK %f\n", elapsed_time(endVideoSequence, endParticleFilter));
    printf("ENTIRE PROGRAM TOOK %f\n", elapsed_time(start, endParticleFilter));

    free(seed);
    free(I);
    return 0;
}
